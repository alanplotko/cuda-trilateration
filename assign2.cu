#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <unistd.h>
#include <utility>

typedef std::pair<double, double> Point;

std::ostream& operator<<(std::ostream& stream, Point& p) {
    return stream << p.first << " " << p.second;
}

// Kernel definition
__global__ void FindPoint(const Point &a, const Point &b, const Point &c, double da, double db, double dc) {
    int i = threadIdx.x;
    float A = (-2 * a.first) + (2 * b.first),
          B = (-2 * a.second) + (2 * b.second),
          C = (da * da) - (db * db) - (a.first * a.first) + (b.first * b.first) - (a.second * a.second) + (b.second * b.second),
          D = (-2 * b.first) + (2 * c.first),
          E = (-2 * b.second) + (2 * c.second),
          F = (db * db) - (dc * dc) - (b.first * b.first) + (c.first * c.first) - (b.second * b.second) + (c.second * c.second);
}

// Set up guard points
void setGuards(std::ifstream &ifs, Point &a, Point &b, Point &c) {
    double x, y;
    ifs >> x >> y;
    a = std::make_pair(x, y);
    ifs >> x >> y;
    b = std::make_pair(x, y);
    ifs >> x >> y;
    c = std::make_pair(x, y);
}

int main(int argc, char* argv[]) {
    // Process flags
    std::string filename;
    int flag;
    opterr = 0;
    while ((flag = getopt(argc, argv, "hi:")) != -1) {
        switch(flag) {
            case 'i':
                filename = optarg;
                break;
            case 'h':
                std::cerr << "Usage: ./gen [-vho] <file-path>\n\n" <<
                             "Options:\n" <<
                             "-h\t\t Show usage string and exit\n" <<
                             "-i <file-path>\t Read input from provided file\n";
                exit(-1);
            case '?':
                if (optopt == 'i') {
                    std::cerr << "Option -" << (char)optopt << " requires an argument.\n";
                } else if (isprint(optopt)) {
                    std::cerr << "Unknown option `-" << (char)optopt << "'.\n";
                } else {
                    std::cerr << "Unknown option character `\\x" << (char)optopt << "'.\n";
                }
                exit(-1);
            default:
                exit(-1);
        }
    }

    // Ensure filename was passed
    if (filename.empty()) {
        std::cerr << "Error: input filename required\n";
        exit(-1);
    }

    // Open input file for reading
    std::ifstream ifs;
    ifs.open(filename.c_str(), std::ios::in);
    if (!ifs.is_open()) {
        std::cerr << "Error: failed to open " << filename << "\n";
        exit(-1);
    }

    // Use API to determine U and V values
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << "\n";
    } else {
        for (int i = 0; i < nDevices; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            std::cerr << "Device Number: " << i << "\n";
            std::cerr << "  Device name: " << prop.name << "\n";
            std::cerr << "  Memory Clock Rate (KHz): " << prop.memoryClockRate <<  "\n";
            std::cerr << "  Memory Bus Width (bits): " << prop.memoryBusWidth <<  "\n";
            std::cerr << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << "\n";
        }
    }

    // Set up guard points
    Point a, b, c, p;
    setGuards(ifs, a, b, c);

    // Read in distances and determine point p
    double da, db, dc;
    while (true) {
        ifs >> da >> db >> dc;

        // Kernel invocation with N threads
        FindPoint<<<1, 1>>>(a, b, c, da, db, dc);

        if(ifs.eof()) break;
    }

    ifs.close();
    return 0;
}
